#include "hip/hip_runtime.h"
/*
 * @author: Zhexuan Gu
 * @Date: 2024-11-02 14:07:25
 * @LastEditTime: 2024-11-05 12:44:59
 * @FilePath: /LearnCudaProgramming/OTMatrixVec.cu
 * @Description: Please implement
 */
#include <torch/extension.h>
// default to deal with square matrix first 
// Grid Layout    dim3 blocks(3, 2m)
// Block Layout   dim3 threads(mn, mn) = (m^2, m^2)
__global__ void OTMVSq(double* v, double* o, int m, const int widthsplit, const int heightsplit){
  // Block index
  int bx = blockIdx.x;
  int by = blockIdx.y;

  // Thread index
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Block dim
  int dx = blockDim.x;
  int dy = blockDim.y;

  // map each thread to sparse matrix entry index
  int Arow = dy * by + ty;
  int Acol = dx * bx + tx;

  if (bx < widthsplit && by < heightsplit) {
    // A1 case
    if(Acol >= Arow * m && Acol < (Arow + 1) * m) {
      atomicAdd(o + Arow, v[Acol]);
    }
  }
  else if (by < heightsplit && bx >= widthsplit){
    // A2 case
    int offset = widthsplit * dx;
    if(((Acol - offset) % (m * m)) == Arow) {
      atomicAdd(o + Arow, -1 * v[Acol]);
    }
  }
  else if (by >= heightsplit && by < 2 * heightsplit && bx < widthsplit) {
    // A3 case 
    int relative_Arow = Arow - heightsplit * dy;
    int subblockid = relative_Arow / m;
    if(Acol >= subblockid * m * m && Acol < (subblockid + 1) * m * m && Acol % m == relative_Arow % m) {
      atomicAdd(o + Arow, v[Acol]);
    }
  }
  else if (by >= 2 * heightsplit && bx >= widthsplit){
    // A4 case
    int relative_Arow = Arow - heightsplit * dy * 2;
    int relative_Acol = Acol - widthsplit * dx;
    int subblockid = relative_Arow / m;
    if(relative_Acol >= subblockid * m * m && relative_Acol < (subblockid + 1) * m * m && relative_Acol % m == relative_Arow % m) {
      atomicAdd(o + Arow, v[Acol]);
    }
  }
}

void OTMVSq_launcher(torch::Tensor& vec, torch::Tensor& out, int vecSize) {
    // Note that we only consider square matrix case
    // additionally, we mainly consider the 32 multiple case
    const int sparseArows = vecSize * vecSize * 3;
    const int sparseAcols = 2 * vecSize * vecSize * vecSize;
    int blockdim = 32;
    dim3 gridSize(sparseAcols / blockdim, sparseArows / blockdim);
    dim3 blockSize(blockdim, blockdim);
    const int widthsplit = vecSize * vecSize * vecSize / blockdim;
    const int heightsplit = vecSize * vecSize / blockdim;
    OTMVSq<<<gridSize, blockSize>>>(vec.data_ptr<double>(), out.data_ptr<double>(), vecSize, widthsplit, heightsplit);
}

